#include "hip/hip_runtime.h"
/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

__global__
void gpu_computation(double* input, double* output, int length);

// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */

    /*----- What I did -----*/
    const long SIZE = length * length * sizeof(double);
    double* gpu_input;
    double* gpu_output;
    dim3 threadsPerBlock(32,32);
    dim3 nbBlocks(length / threadsPerBlock.x + 1, length / threadsPerBlock.y + 1);
    const long PADDED_SIZE = (nbBlocks.x+1) * threadsPerBlock.x * (nbBlocks.y+1) * threadsPerBlock.y * sizeof(double); //+1 to avoid going out of the input
    hipSetDevice(0);
    if(hipMalloc((void**)&gpu_input, PADDED_SIZE) != hipSuccess){
        cerr << "Error allocating input" << endl;
    }
    if(hipMalloc((void**)&gpu_output, PADDED_SIZE) != hipSuccess){
        cerr << "Error allocating output" << endl;
    }
    /*----------------------*/

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */

    /*----- What I did -----*/
    if(hipMemcpy(gpu_input, input, SIZE, hipMemcpyHostToDevice) != hipSuccess){
        cerr << "Error copying input to gpu" << endl;
    }

    if(hipMemcpy(gpu_output, output, SIZE, hipMemcpyHostToDevice) != hipSuccess){
        cerr << "Error copying output to gpu" << endl;
    }
    /*----------------------*/

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */

    /*----- What I did -----*/
    for(int iter(0); iter < iterations; iter++){
        if(iter%2){ 
            gpu_computation <<< nbBlocks, threadsPerBlock >>> (gpu_output, gpu_input, length);
        }
        else{
            gpu_computation <<< nbBlocks, threadsPerBlock >>> (gpu_input, gpu_output, length);
        }
        hipDeviceSynchronize();
    }
    /*----------------------*/

    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */

    /*----- What I did -----*/
    if(iterations%2==0)
    {
        if(hipMemcpy(output, gpu_input, SIZE, hipMemcpyDeviceToHost) != hipSuccess){
            cerr << "failed to retrieve gpu_input" << endl;
        }
    }
    else{
        if(hipMemcpy(output, gpu_output, SIZE, hipMemcpyDeviceToHost) != hipSuccess){
            cerr << "failed to retrieve gpu_output" << endl;
        }
    }
    /*----------------------*/

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */

    /*----- What I did -----*/
    hipFree(&gpu_input);
    hipFree(&gpu_output);
    /*----------------------*/

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}

/*With & instead of branches single kernel*/
//DOES NOT PRODUCE CORRECT RESULT : need to extend ones
__global__
void gpu_computation(double* input, double* output, int length){
    int x_glob = (blockIdx.x * blockDim.x) + threadIdx.x + 1;   //+1 to avoid first column
    int y_glob = (blockIdx.y * blockDim.y) + threadIdx.y + 1;   //+1 to avoid first row
    int element_id = (y_glob * length) + x_glob;
    bool isCenter = ((x_glob == length/2-1) || (x_glob == length/2)) && ((y_glob == length/2-1) || (y_glob == length/2));
    bool isBorder = x_glob == 0 || y_glob == 0 || x_glob >= length - 1 || y_glob >= length-1;

    double temp((input[(y_glob-1)*(length)+(x_glob-1)] +
                 input[(y_glob-1)*(length)+(x_glob)]   +
                 input[(y_glob-1)*(length)+(x_glob+1)] +
                 input[(y_glob)*(length)+(x_glob-1)]   +
                 input[(y_glob)*(length)+(x_glob)]     +
                 input[(y_glob)*(length)+(x_glob+1)]   +
                 input[(y_glob+1)*(length)+(x_glob-1)] +
                 input[(y_glob+1)*(length)+(x_glob)]   +
                input[(y_glob+1)*(length)+(x_glob+1)]) / 9);
    output[element_id] = (isCenter & 1000) + (isBorder & 0) + (!(isCenter || isBorder) & temp);
}
/**/

/*With mult instead of branches single kernel
__global__
void gpu_computation(double* input, double* output, int length){
    int x_glob = (blockIdx.x * blockDim.x) + threadIdx.x + 1;   //+1 to avoid first column
    int y_glob = (blockIdx.y * blockDim.y) + threadIdx.y + 1;   //+1 to avoid first row
    int element_id = (y_glob * length) + x_glob;
    int isCenter = ((x_glob == length/2-1) || (x_glob == length/2)) && ((y_glob == length/2-1) || (y_glob == length/2));
    int isBorder = x_glob == 0 || y_glob == 0 || x_glob >= length - 1 || y_glob >= length-1;

    output[element_id] = isCenter * 1000 + isBorder * 0 + !(isCenter || isBorder) *
                                            ((input[(y_glob-1)*(length)+(x_glob-1)] +
                                            input[(y_glob-1)*(length)+(x_glob)]   +
                                            input[(y_glob-1)*(length)+(x_glob+1)] +
                                            input[(y_glob)*(length)+(x_glob-1)]   +
                                            input[(y_glob)*(length)+(x_glob)]     +
                                            input[(y_glob)*(length)+(x_glob+1)]   +
                                            input[(y_glob+1)*(length)+(x_glob-1)] +
                                            input[(y_glob+1)*(length)+(x_glob)]   +
                                            input[(y_glob+1)*(length)+(x_glob+1)] ) / 9);
}
*/

/*Without branches single kernel
__global__
void gpu_computation(double* input, double* output, int length){
    int x_glob = (blockIdx.x * blockDim.x) + threadIdx.x + 1;   //+1 to avoid first column
    int y_glob = (blockIdx.y * blockDim.y) + threadIdx.y + 1;   //+1 to avoid first row
    int element_id = (y_glob * length) + x_glob;
    int isCenter = ((x_glob == length/2-1) || (x_glob == length/2)) && ((y_glob == length/2-1) || (y_glob == length/2));
    int isBorder = x_glob == 0 || y_glob == 0 || x_glob >= length - 1 || y_glob >= length-1;

    output[element_id] = isCenter ? 1000 : (isBorder ? 0 : (input[(y_glob-1)*(length)+(x_glob-1)] +
                                            input[(y_glob-1)*(length)+(x_glob)]   +
                                            input[(y_glob-1)*(length)+(x_glob+1)] +
                                            input[(y_glob)*(length)+(x_glob-1)]   +
                                            input[(y_glob)*(length)+(x_glob)]     +
                                            input[(y_glob)*(length)+(x_glob+1)]   +
                                            input[(y_glob+1)*(length)+(x_glob-1)] +
                                            input[(y_glob+1)*(length)+(x_glob)]   +
                                            input[(y_glob+1)*(length)+(x_glob+1)] ) / 9);
}
*/

/*With branches single kernel
__global__
void gpu_computation(double* input, double* output, int length){
    int x_glob = (blockIdx.x * blockDim.x) + threadIdx.x + 1;   //+1 to avoid first column
    int y_glob = (blockIdx.y * blockDim.y) + threadIdx.y + 1;   //+1 to avoid first row
    int element_id = (y_glob * length) + x_glob;
    if ( ((x_glob == length/2-1) || (x_glob == length/2)) && ((y_glob == length/2-1) || (y_glob == length/2)) 
        || x_glob >= length - 1 || y_glob >= length-1)
    {
        return;
    }
    output[element_id] = (input[(y_glob-1)*(length)+(x_glob-1)] +
                                            input[(y_glob-1)*(length)+(x_glob)]   +
                                            input[(y_glob-1)*(length)+(x_glob+1)] +
                                            input[(y_glob)*(length)+(x_glob-1)]   +
                                            input[(y_glob)*(length)+(x_glob)]     +
                                            input[(y_glob)*(length)+(x_glob+1)]   +
                                            input[(y_glob+1)*(length)+(x_glob-1)] +
                                            input[(y_glob+1)*(length)+(x_glob)]   +
                                            input[(y_glob+1)*(length)+(x_glob+1)] ) / 9;
}
*/