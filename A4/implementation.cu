/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */

    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}